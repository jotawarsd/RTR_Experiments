#include "hip/hip_runtime.h"
//Header Files
#include <stdio.h>
#include <stdlib.h>
#include "OGLCU.h"
#include <C:/glew/include/GL/glew.h>	//this must be above gl.h

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#pragma comment(lib, "cudart.lib")

FILE *gpFile = NULL;

//sine wave variables
const unsigned int meshWidth = 8192;
const unsigned int meshHeight = 8192;

#define MYARRAYSIZE meshWidth * meshHeight * 4

float pos[meshWidth][meshHeight][4];

GLuint vbo_gpu;
hipError_t cudaResult;

float animationTime = 0.0f;

//CUDA Kernel
__global__ void sineWaveKernel(float4 *position, unsigned int width, unsigned int height, float time)
{
	//code
	unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	unsigned int j = (blockIdx.y * blockDim.y) + threadIdx.y;

	float u = (float)i / (float)width;
	float v = (float)j / (float)height;

	u = u * 2.0f - 1.0f;
	v = v * 2.0f - 1.0f;

	float frequency = 4.0f;
	float w = sinf((u * frequency) + time) * cosf((v * frequency) + time) * 0.5f;

	position[(j * width) + i] = make_float4(u, w, v, 1.0f);
}

void cudaInit(GLuint vbo, struct hipGraphicsResource * graphicsResource)
{
	if (fopen_s(&gpFile, "cudaLog.txt", "w") == 0)
		fprintf(gpFile, "Log File for CUDA Created Successfully!\n");

	//CUDA Initialization
	int dev_count = 0;
	cudaResult = hipGetDeviceCount(&dev_count);
	if (cudaResult != hipSuccess)
	{
		fprintf(gpFile, "Cuda device count failed!!\n");
		exit(EXIT_FAILURE);
	}
	else if (dev_count == 0)
	{
		fprintf(gpFile, "No CUDA Supported devices\n");
		exit(EXIT_FAILURE);
	}
	fprintf(gpFile, "Number of CUDA Supported devices: %d\n", dev_count);
	
	//select cuda supported device
	hipSetDevice(0);	//selecting the default 0th cuda supported device

	//create cuda-opengl inter-operability resource
	cudaResult = hipGraphicsGLRegisterBuffer(&graphicsResource, vbo, cudaGraphicsMapFlagsWriteDiscard);
	if (cudaResult != hipSuccess)
	{
		fprintf(gpFile, "hipGraphicsGLRegisterBuffer() failed!!\n");
		exit(EXIT_FAILURE);
	}
}

void cudaSineWave(struct hipGraphicsResource * graphicsResource, int width, int height, float time)
{
	//variable declarations
	float4 *pPos = NULL;
	size_t numBytes;
	//cuda code
	//map cuda graphics resource
	cudaResult = hipGraphicsMapResources(1, &graphicsResource, 0);
	if (cudaResult != hipSuccess)
	{
		fprintf(gpFile, "hipGraphicsMapResources() failed!!\n");
		exit(EXIT_FAILURE);
	}

	//ask cuda to give mapped pointer of mapped resource
	cudaResult = hipGraphicsResourceGetMappedPointer((void **)&pPos, &numBytes, graphicsResource);
	if (cudaResult != hipSuccess)
	{
		fprintf(gpFile, "hipGraphicsResourceGetMappedPointer() failed!!\n");
		exit(EXIT_FAILURE);
	}

	//call cuda sineWave Kernel
	dim3 block(8, 8, 1);
	dim3 grid(width, height, 1);

	sineWaveKernel <<<grid, block>>> (pPos, width, height, time);

	//unmap given mapped pointer to use on cpu
	cudaResult = hipGraphicsUnmapResources(1, &graphicsResource, 0);
	if (cudaResult != hipSuccess)
	{
		fprintf(gpFile, "hipGraphicsUnmapResources() failed!!\n");
		exit(EXIT_FAILURE);
	}
}
