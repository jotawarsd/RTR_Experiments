#include "hip/hip_runtime.h"
//Header Files
#include <windows.h>
#include "OGL.h"
#include <stdio.h>
#include <stdlib.h>

//OpenGL Header files
#include <C:/glew/include/GL/glew.h>	//this must be above gl.h
#include <GL/gl.h>
#include "vmath.h"
using namespace vmath;

//CUDA headers
#include <hip/hip_runtime.h>

//macros
#define X GetSystemMetrics(SM_CXSCREEN)
#define Y GetSystemMetrics(SM_CYSCREEN)
#define WIN_WIDTH 800
#define WIN_HEIGHT 600
#define MANDELBROT_WIDTH 6000
#define MANDELBROT_HEIGHT 6000

//OpenGL Libraries
#pragma comment(lib, "glew32.lib")
#pragma comment(lib, "OpenGL32.lib")

//Global Function Declarations
LRESULT CALLBACK WndProc(HWND, UINT, WPARAM, LPARAM);

//Global variable declarations
HWND ghwnd = NULL;
HDC ghdc = NULL;
BOOL gbFullscreen = FALSE;
FILE *gpFile = NULL;
BOOL gbActiveWindow = FALSE;
HGLRC ghrc = NULL;

//Programmable pipeline related global variables
GLuint shaderProgramObject;

enum
{
	SDJ_ATTRIBUTE_POSITION = 0,
	SDJ_ATTRIBUTE_COLOR,
	SDJ_ATTRIBUTE_NORMAL,
	SDJ_ATTRIBUTE_TEXTURE0
};

GLuint vao;
GLuint vbo_position;
GLuint vbo_texcoord;

GLint CheckerBoard[MANDELBROT_WIDTH][MANDELBROT_HEIGHT][4];
GLubyte Mandelbrot_Host[MANDELBROT_WIDTH][MANDELBROT_HEIGHT][4];
unsigned char *Mandelbrot_Device = NULL;
GLuint texture_Mandelbrot;

int numIterations = 0;
BOOL iterChanged = TRUE;

GLuint mvpMatrixUniform;
GLuint textureSamplerUniform;

mat4 perspectiveProjectionMatrix;

//cuda kernel
__global__ void mandelbrotKernel(unsigned char *pixel, int width, int height, int maxIter)
{
	//code
	unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	unsigned int j = (blockIdx.y * blockDim.y) + threadIdx.y;

	float u = (float)i / (float)width;
	float v = (float)j / (float)height;

	u = u * 3.0f - 2.4f;
	v = v * 3.0f - 1.5f;

	float x = 0.0f, y = 0.0f;
	double iter = 0.0;
	while(x * x + y * y <= 4.0f && iter < maxIter)
	{
		float xtemp = x * x - y * y + u;
		y = 2.0f * x * y + v;
		x = xtemp;
		iter++;
	}
	iter /= 10.0f;

	pixel[(j * width * 4) + i * 4 + 0] = iter * 0.3 * 255;
	pixel[(j * width * 4) + i * 4 + 1] = iter * 0.5 * 255;
	pixel[(j * width * 4) + i * 4 + 2] = iter * 255;
	pixel[(j * width * 4) + i * 4 + 3] = 0;
}

//Entry Point Function
int WINAPI WinMain(HINSTANCE hInstance, HINSTANCE hPrevInstance, LPSTR lpszCmdLine, int iCmdShow)
{
	//Function Declarations
	int initialize(void);
	void display(void);
	//void update(void);	-----no animation. hence update call not required
	void uninitialize(void);

	//Variable Declarations
	WNDCLASSEX wndclass;
	HWND hwnd;
	MSG msg;
	TCHAR szAppName[] = TEXT("MyWindow");
	BOOL bDone = FALSE;
	int iRetVal = 0;

	//code
	if (fopen_s(&gpFile, "Log.txt", "w") != 0)
	{
		MessageBox(NULL, TEXT("Creation of Log File Failed. Exiting now!"), TEXT("File I/O Error"), MB_OK | MB_ICONEXCLAMATION);
		exit(0);
	}
	else
		fprintf(gpFile, "Log File Created Successfully!\n");

	//Initialization of WNDCLASSEX structure
	wndclass.cbSize = sizeof(WNDCLASSEX);    //not included in WNDCLASS (only WNDCLASSEX)
	wndclass.style = CS_HREDRAW | CS_VREDRAW | CS_OWNDC;
	wndclass.cbClsExtra = 0;
	wndclass.cbWndExtra = 0;
	wndclass.lpfnWndProc = WndProc;
	wndclass.hInstance = hInstance;
	wndclass.hbrBackground = (HBRUSH)GetStockObject(BLACK_BRUSH);	//Client area color (Canvas)
	wndclass.hIcon = LoadIcon(hInstance, MAKEINTRESOURCE(MYICON));
	wndclass.hCursor = LoadCursor(NULL, IDC_ARROW);
	wndclass.lpszClassName = szAppName;
	wndclass.lpszMenuName = NULL;
	wndclass.hIconSm = LoadIcon(hInstance, MAKEINTRESOURCE(MYICON));    //not included in WNDCLASS

	//Registering Above class
	RegisterClassEx(&wndclass);

	//Creating the window
	hwnd = CreateWindowEx(
		WS_EX_APPWINDOW,																		//Extended style for fullscreen (window that remains above taskbar always)
		szAppName,																				//Name of Wnd class
		TEXT("Shaunak Jotawar OpenGL"),															//Title bar text
		WS_OVERLAPPEDWINDOW | WS_CLIPCHILDREN | WS_CLIPSIBLINGS | WS_VISIBLE,					//style of window 
																								//(3 additional styles ensure fullscreen window stays in focus above all children and sibling windows)
		X / 2 - (WIN_WIDTH / 2),																//x coordinate of top left corner
		Y / 2 - (WIN_HEIGHT / 2),																//y coordinate of top left corner
		WIN_WIDTH,																				//width
		WIN_HEIGHT,																				//height
		NULL,																					//parent window (NULL = Desktop)
		NULL,																					//Handle to Menu (NULL = no menu)
		hInstance,																				//handle to instance
		NULL);																					//Extra info for window

	ghwnd = hwnd;

	//Initialize
	iRetVal = initialize();
	if (iRetVal == -1)
	{
		fprintf(gpFile, "Choose Pixel Format failed\n");
		uninitialize();
	}
	else if (iRetVal == -2)
	{
		fprintf(gpFile, "Set Pixel Format failed\n");
		uninitialize();
	}
	else if (iRetVal == -3)
	{
		fprintf(gpFile, "Create OpenGL context failed\n");
		uninitialize();
	}
	else if (iRetVal == -4)
	{
		fprintf(gpFile, "Making OpenGL as current context failed\n");
		uninitialize();
	}
	else if (iRetVal == -5)
	{
		fprintf(gpFile, "Initialization of GLEW failed\n");
	}
	else if (iRetVal == -6)
	{
		fprintf(gpFile, "LoadGLTexture for Texture_Stone Failed\n");
		uninitialize();
	}
	else
		fprintf(gpFile, "Initialization successful!\n");

	//Show Window
	ShowWindow(hwnd, iCmdShow);

	//Foregrounding and focusing the window
	SetForegroundWindow(hwnd);
	SetFocus(hwnd);

	//Game loop
	while (bDone == FALSE)
	{
		if (PeekMessage(&msg, NULL, 0, 0, PM_REMOVE))
		{
			if (msg.message == WM_QUIT)
				bDone = TRUE;
			else
			{
				TranslateMessage(&msg);
				DispatchMessage(&msg);
			}
		}
		else
		{
			if (gbActiveWindow == TRUE)
			{
				//Rendering the scene
				display();

				//updating the scene
				//update();
			}
		}
	}

	uninitialize();
	return ((int)msg.wParam);
}

//Callback Function
LRESULT CALLBACK WndProc(HWND hwnd, UINT iMsg, WPARAM wParam, LPARAM lParam)
{
	//Function Prototypes
	void ToggleFullscreen(void);
	void resize(int, int);

	//code
	switch (iMsg)
	{
	case WM_SETFOCUS:
		gbActiveWindow = TRUE;
		break;

	case WM_KILLFOCUS:
		gbActiveWindow = FALSE;
		break;

	case WM_ERASEBKGND:
		return 0;

	case WM_SIZE:
		resize(LOWORD(lParam), HIWORD(lParam));
		break;

	case WM_CHAR:
		switch (wParam)
		{
		case 'F':
		case 'f':
			ToggleFullscreen();
			break;
		default:
			break;
		}
		break;

	case WM_KEYDOWN:
		switch (wParam)
		{
		case VK_UP:
			numIterations += 10;
			iterChanged = TRUE;
			break;
		case VK_DOWN:
			numIterations -= 10;
			iterChanged = TRUE;
			break;
		case 27:
			DestroyWindow(hwnd);
			break;
		default:
			break;
		}
		break;

	case WM_CLOSE:
		DestroyWindow(hwnd);
		break;

	case WM_DESTROY:
		PostQuitMessage(0);
		break;

	default:
		break;
	}

	return DefWindowProc(hwnd, iMsg, wParam, lParam);
}

void ToggleFullscreen(void)
{
	//variable declarations
	static DWORD dwStyle;
	static WINDOWPLACEMENT wp;
	MONITORINFO mi;
	LONG left, top, width, height;

	//code
	wp.length = sizeof(WINDOWPLACEMENT);

	if (gbFullscreen == FALSE)
	{
		dwStyle = GetWindowLong(ghwnd, GWL_STYLE);
		if (dwStyle & WS_OVERLAPPEDWINDOW)
		{
			mi.cbSize = sizeof(MONITORINFO);

			if (GetWindowPlacement(ghwnd, &wp) && GetMonitorInfo(MonitorFromWindow(ghwnd, MONITORINFOF_PRIMARY), &mi))
			{
				SetWindowLong(ghwnd, GWL_STYLE, dwStyle & ~WS_OVERLAPPEDWINDOW);

				left = mi.rcMonitor.left;
				top = mi.rcMonitor.top;
				width = mi.rcMonitor.right - mi.rcMonitor.left;
				height = mi.rcMonitor.bottom - mi.rcMonitor.top;

				SetWindowPos(ghwnd, HWND_TOP, left, top, width, height, SWP_NOZORDER | SWP_FRAMECHANGED);
			}

			ShowCursor(FALSE);
			gbFullscreen = TRUE;
		}
	}
	else
	{
		SetWindowLong(ghwnd, GWL_STYLE, dwStyle | WS_OVERLAPPEDWINDOW);
		SetWindowPlacement(ghwnd, &wp);
		SetWindowPos(ghwnd, HWND_TOP, 0, 0, 0, 0, SWP_NOMOVE | SWP_NOOWNERZORDER | SWP_NOZORDER | SWP_NOSIZE | SWP_FRAMECHANGED);

		ShowCursor(TRUE);
		gbFullscreen = FALSE;
	}
}

int initialize(void)
{
	//Function Declarations
	void uninitialize(void);
	void resize(int, int);
	void printGLInfo(void);
	void loadGLTexture(void);

	//variable declarations
	PIXELFORMATDESCRIPTOR pfd;
	int iPixelFormatIndex = 0;

	//code
	ZeroMemory(&pfd, sizeof(PIXELFORMATDESCRIPTOR));

	//initializing PIXELFORMATDESCRIPTOR structure
	pfd.nSize = sizeof(PIXELFORMATDESCRIPTOR);
	pfd.nVersion = 1;
	pfd.dwFlags = PFD_DRAW_TO_WINDOW | PFD_SUPPORT_OPENGL | PFD_DOUBLEBUFFER;
	pfd.iPixelType = PFD_TYPE_RGBA;
	pfd.cColorBits = 32;
	pfd.cRedBits = 8;
	pfd.cGreenBits = 8;
	pfd.cBlueBits = 8;
	pfd.cAlphaBits = 8;
	pfd.cDepthBits = 32;

	//get dc
	ghdc = GetDC(ghwnd);

	//Choose Pixel Format
	iPixelFormatIndex = ChoosePixelFormat(ghdc, &pfd);
	if (iPixelFormatIndex == 0)
		return -1;

	//Set the chosen pixel format
	if (SetPixelFormat(ghdc, iPixelFormatIndex, &pfd) == FALSE)
		return -2;

	//Create OpenGL rendering context
	ghrc = wglCreateContext(ghdc);
	if (ghrc == NULL)
		return -3;

	//Make the rendering context as current context
	if (wglMakeCurrent(ghdc, ghrc) == FALSE)
		return -4;

	//GLEW initialization
	if (glewInit() != GLEW_OK)
		return -5;

	//Vertex Shader
	const GLchar* vertexShaderSourceCode =
		"#version 460 core" \
		"\n" \
		"in vec4 a_position;" \
		"in vec2 a_texcoord;" \
		"uniform mat4 u_mvpMatrix;" \
		"out vec2 a_texcoord_out;" \
		"void main(void)" \
		"{" \
		"gl_Position = u_mvpMatrix * a_position;" \
		"a_texcoord_out = a_texcoord;" \
		"}";

	GLuint vertexShaderObject = glCreateShader(GL_VERTEX_SHADER);
	glShaderSource(vertexShaderObject, 1, (const GLchar**)&vertexShaderSourceCode, NULL);
	glCompileShader(vertexShaderObject);

	GLint status;
	GLint infoLogLength;
	char *log = NULL;

	glGetShaderiv(vertexShaderObject, GL_COMPILE_STATUS, &status);
	if (status == GL_FALSE)
	{
		glGetShaderiv(vertexShaderObject, GL_INFO_LOG_LENGTH, &infoLogLength);
		if (infoLogLength > 0)
		{
			log = (char *)malloc(infoLogLength);
			if (log != NULL)
			{
				GLsizei written;
				glGetShaderInfoLog(vertexShaderObject, infoLogLength, &written, log);
				fprintf(gpFile, "\n\nVertex Shader Compilation Log : %s\n\n", log);
				free(log);
				log = NULL;
				uninitialize();
			}
		}
	}

	//Fragment Shader
	const GLchar* fragmentShaderSourceCode =
		"#version 460 core" \
		"\n" \
		"in vec2 a_texcoord_out;" \
		"uniform sampler2D u_textureSampler;" \
		"out vec4 FragColor;" \
		"void main(void)" \
		"{" \
		"FragColor = texture(u_textureSampler, a_texcoord_out);" \
		"}";

	GLuint fragmentShaderObject = glCreateShader(GL_FRAGMENT_SHADER);
	glShaderSource(fragmentShaderObject, 1, (const GLchar**)&fragmentShaderSourceCode, NULL);
	glCompileShader(fragmentShaderObject);

	status = 0;
	infoLogLength = 0;
	log = NULL;

	glGetShaderiv(fragmentShaderObject, GL_COMPILE_STATUS, &status);
	if (status == GL_FALSE)
	{
		glGetShaderiv(fragmentShaderObject, GL_INFO_LOG_LENGTH, &infoLogLength);
		if (infoLogLength > 0)
		{
			log = (char*)malloc(infoLogLength);
			if (log != NULL)
			{
				GLsizei written;
				glGetShaderInfoLog(fragmentShaderObject, infoLogLength, &written, log);
				fprintf(gpFile, "\n\nFragment Shader Compilation Log : %s", log);
				free(log);
				log = NULL;
				uninitialize();
			}
		}
	}

	//Shader Program object
	shaderProgramObject = glCreateProgram();
	glAttachShader(shaderProgramObject, vertexShaderObject);
	glAttachShader(shaderProgramObject, fragmentShaderObject);

	glBindAttribLocation(shaderProgramObject, SDJ_ATTRIBUTE_POSITION, "a_position");
	glBindAttribLocation(shaderProgramObject, SDJ_ATTRIBUTE_TEXTURE0, "a_texcoord");

	glLinkProgram(shaderProgramObject);


	//ERROR Checking
	status = 0;
	infoLogLength = 0;
	log = NULL;

	glGetProgramiv(shaderProgramObject, GL_LINK_STATUS, &status);
	if (status == GL_FALSE)
	{
		glGetProgramiv(shaderProgramObject, GL_INFO_LOG_LENGTH, &infoLogLength);
		if (infoLogLength > 0)
		{
			log = (char*)malloc(infoLogLength);
			if (log != NULL)
			{
				GLsizei written;
				glGetProgramInfoLog(shaderProgramObject, infoLogLength, &written, log);
				fprintf(gpFile, "\n\nShader Program Link log : %s", log);
				free(log);
				log = NULL;
				uninitialize();
			}
		}
	}

	//post link steps
	mvpMatrixUniform = glGetUniformLocation(shaderProgramObject, "u_mvpMatrix");
	textureSamplerUniform = glGetUniformLocation(shaderProgramObject, "u_textureSampler");

	//declarations of vertex data arrays
	const GLfloat position[] = 
	{
		1.0f, 1.0f, 0.0f,
		-1.0f, 1.0f, 0.0f,
		-1.0f, -1.0f, 0.0f,
		1.0f, -1.0f, 0.0f
	};

	const GLfloat texCoords[] =
	{
		1.0f, 1.0f,
		0.0f, 1.0f,
		0.0f, 0.0f,
		1.0f, 0.0f
	};

	//vao and vbo related code
	//vertex array object
	glGenVertexArrays(1, &vao);
	glBindVertexArray(vao);

	//vertex buffer object for ___position___
	glGenBuffers(1, &vbo_position);
	glBindBuffer(GL_ARRAY_BUFFER, vbo_position);

	glBufferData(GL_ARRAY_BUFFER, sizeof(position), position, GL_STATIC_DRAW);
	glVertexAttribPointer(SDJ_ATTRIBUTE_POSITION, 3, GL_FLOAT, GL_FALSE, 0, NULL);
	glEnableVertexAttribArray(SDJ_ATTRIBUTE_POSITION);

	glBindBuffer(GL_ARRAY_BUFFER, 0);

	//vertex buffer object for ___texcoords___
	glGenBuffers(1, &vbo_texcoord);
	glBindBuffer(GL_ARRAY_BUFFER, vbo_texcoord);

	glBufferData(GL_ARRAY_BUFFER, sizeof(texCoords), texCoords, GL_STATIC_DRAW);
	glVertexAttribPointer(SDJ_ATTRIBUTE_TEXTURE0, 2, GL_FLOAT, GL_FALSE, 0, NULL);
	glEnableVertexAttribArray(SDJ_ATTRIBUTE_TEXTURE0);

	glBindBuffer(GL_ARRAY_BUFFER, 0);

	//unbind __vao__
	glBindVertexArray(0);

	//Depth enabling and clear color functions
	glClearDepth(1.0f);
	glEnable(GL_DEPTH_TEST);
	glDepthFunc(GL_LEQUAL);

	//load texture
	loadGLTexture();

	glEnable(GL_TEXTURE_2D);

	//clear color
	glClearColor(0.15f, 0.15f, 0.15f, 1.0f);

	//initializing perspective projection matrix
	perspectiveProjectionMatrix = mat4::identity();
	resize(WIN_WIDTH, WIN_HEIGHT);

	return 0;
}

void loadGLTexture(void)
{
	//Function Declarations
	void makeMandelbrot(void);

	//code
	makeMandelbrot();
	glPixelStorei(GL_UNPACK_ALIGNMENT, 1);

	glGenTextures(1, &texture_Mandelbrot);
	glBindTexture(GL_TEXTURE_2D, texture_Mandelbrot);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);

	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, MANDELBROT_WIDTH, MANDELBROT_HEIGHT, 0, GL_RGBA, GL_UNSIGNED_BYTE, Mandelbrot_Host);
	glTexEnvf(GL_TEXTURE_ENV, GL_TEXTURE_ENV_MODE, GL_REPLACE);

	glBindTexture(GL_TEXTURE_2D, 0);
}

void makeMandelbrot(void)
{
	//function prototypes
	void uninitialize(void);

	//variable declarations
	hipError_t result = hipSuccess;

	//code
	//Device memory allocation
	result = hipMalloc((void **)&Mandelbrot_Device, MANDELBROT_WIDTH * MANDELBROT_HEIGHT * 4 * sizeof(int));
	if (result != hipSuccess)
	{
		fprintf(gpFile, "\nDevice Memory Allocation Failed for Mandelbrot_Device\n");
		uninitialize();
	}

	//kernel launch
	dim3 block(32, 32, 1);
	dim3 grid(MANDELBROT_WIDTH / block.x, MANDELBROT_HEIGHT / block.y, 1);

	//kernel call
	mandelbrotKernel <<<grid, block>>> (Mandelbrot_Device, MANDELBROT_WIDTH, MANDELBROT_HEIGHT, numIterations);

	//copy result from device to host
	result = hipMemcpy(Mandelbrot_Host, Mandelbrot_Device, MANDELBROT_WIDTH * MANDELBROT_HEIGHT * 4 * sizeof(char), hipMemcpyDeviceToHost);
	if (result != hipSuccess)
	{
		fprintf(gpFile, "\nDevice to Host Memory Copy Failed for Mandelbrot_Host\n");
		uninitialize();
	}
}

void resize(int width, int height)
{
	//code
	if (height == 0)
		height = 1;

	glViewport(0, 0, (GLsizei)width, (GLsizei)height);

	perspectiveProjectionMatrix = vmath::perspective(
		45.0, (GLfloat)width / (GLfloat)height, 0.1f, 100.0f
	);
}

void display(void)
{
	//code
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	//use shader program object
	glUseProgram(shaderProgramObject);

	//transformations
	mat4 translationMatrix = mat4::identity();
	mat4 modelViewMatrix = mat4::identity();
	mat4 modelViewProjectionMatrix = mat4::identity();
	
	translationMatrix = translate(0.0f, 0.0f, -2.0f);

	modelViewMatrix = translationMatrix;
	modelViewProjectionMatrix = perspectiveProjectionMatrix * modelViewMatrix;

	glUniformMatrix4fv(mvpMatrixUniform, 1, GL_FALSE, modelViewProjectionMatrix);

	//texture binding
	if (iterChanged)
	{
		loadGLTexture();
	
		TCHAR str[255];
		wsprintf(str, TEXT("Mandelbrot Set : %d iterations"), numIterations);
		SetWindowText(ghwnd, str);
		
		iterChanged = FALSE;
	}
	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_2D, texture_Mandelbrot);
	glUniform1i(textureSamplerUniform, 0);

	//Draw
	glBindVertexArray(vao);
	glDrawArrays(GL_TRIANGLE_FAN, 0, 4);
	glBindVertexArray(0);

	//unuse the shader program object
	glUseProgram(0);

	SwapBuffers(ghdc);
}

//void update(void)
//{
//	//code
//}

void uninitialize(void)
{
	//Function declarations
	void ToggleFullscreen(void);

	//code
	if (gbFullscreen)
	{
		ToggleFullscreen();
	}

	if (Mandelbrot_Device)
	{
		hipFree(Mandelbrot_Device);
		Mandelbrot_Device = NULL;
	}

	//deletion and uninitialization of texture
	if (texture_Mandelbrot)
	{
		glDeleteTextures(1, &texture_Mandelbrot);
		texture_Mandelbrot = 0;
	}

	//deletion and uninitialization of vbo
	if (vbo_texcoord)
	{
		glDeleteBuffers(1, &vbo_texcoord);
		vbo_texcoord = 0;
	}

	if (vbo_position)
	{
		glDeleteBuffers(1, &vbo_position);
		vbo_position = 0;
	}

	if (vao)
	{
		glDeleteVertexArrays(1, &vao);
		vao = 0;
	}

	//shader uninitialization
	if (shaderProgramObject)
	{
		glUseProgram(shaderProgramObject);
		GLsizei numAttachedShaders;
		glGetProgramiv(shaderProgramObject, GL_ATTACHED_SHADERS, &numAttachedShaders);
		GLuint *shaderObjects = NULL;
		shaderObjects = (GLuint *)malloc(numAttachedShaders * sizeof(GLuint));

		//filling empty buffer with shader objects
		glGetAttachedShaders(shaderProgramObject, numAttachedShaders, &numAttachedShaders, shaderObjects);

		for (GLsizei i = 0; i < numAttachedShaders; i++)
		{
			glDetachShader(shaderProgramObject, shaderObjects[i]);
			glDeleteShader(shaderObjects[i]);
			shaderObjects[i] = 0;
		}

		free(shaderObjects);
		shaderObjects = NULL;
		glUseProgram(0);
		glDeleteProgram(shaderProgramObject);
		shaderProgramObject = 0;
	}

	if (wglGetCurrentContext() == ghrc)
	{
		wglMakeCurrent(NULL, NULL);
		fprintf(gpFile, "Current context initialized to NULL\n");
	}

	if (ghrc)
	{
		wglDeleteContext(ghrc);
		ghrc = NULL;
		fprintf(gpFile, "Current context deleted\n");
	}

	if (ghdc)
	{
		ReleaseDC(ghwnd, ghdc);
		ghdc = NULL;
		fprintf(gpFile, "Device context released\n");
	}

	if (ghwnd)
	{
		fprintf(gpFile, "Window destroyed\n");
		DestroyWindow(ghwnd);
		ghwnd = NULL;
	}

	if (gpFile)
	{
		fprintf(gpFile, "Log File closed successfully!\n");
		fclose(gpFile);
		gpFile = NULL;
	}

	exit(0);
}
