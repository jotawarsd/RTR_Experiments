#include "hip/hip_runtime.h"
//Header Files
#include <windows.h>
#include "OGL.h"
#include <stdio.h>
#include <stdlib.h>
#include <C:/glew/include/GL/glew.h>	//this must be above gl.h

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

//OpenGL Header files
#include <GL/gl.h>
#include "vmath.h"
using namespace vmath;

//macros
#define X GetSystemMetrics(SM_CXSCREEN)
#define Y GetSystemMetrics(SM_CYSCREEN)
#define WIN_WIDTH 800
#define WIN_HEIGHT 600

//OpenGL Libraries
#pragma comment(lib, "glew32.lib")
#pragma comment(lib, "OpenGL32.lib")
#pragma comment(lib, "cudart.lib")

//Global Function Declarations
LRESULT CALLBACK WndProc(HWND, UINT, WPARAM, LPARAM);

//Global variable declarations
HWND ghwnd = NULL;
HDC ghdc = NULL;
BOOL gbFullscreen = FALSE;
FILE *gpFile = NULL;
BOOL gbActiveWindow = FALSE;
HGLRC ghrc = NULL;

//Programmable pipeline related global variables
GLuint shaderProgramObject;

enum
{
	SDJ_ATTRIBUTE_POSITION = 0,
	SDJ_ATTRIBUTE_COLOR,
	SDJ_ATTRIBUTE_NORMAL,
	SDJ_ATTRIBUTE_TEXTURE0
};

GLuint vao;
GLuint vbo;
GLuint vbo_gpu;
GLuint vbo_texcoord;

GLuint texture_Mandelbrot;
GLuint mvpMatrixUniform;
GLuint textureSamplerUniform;

mat4 perspectiveProjectionMatrix;

//mandelbrot variables
const unsigned int meshWidth = 1024;
const unsigned int meshHeight = 1024;

#define MYARRAYSIZE meshWidth * meshHeight * 4

float4 *pPos = NULL;
vec4 *pColor = NULL;

hipError_t cudaResult;
BOOL iterChanged = FALSE;
int iterations = 100;

struct hipGraphicsResource *graphicsResource = NULL;

//CUDA Kernel
__global__ void mandelbrotKernel(float4 *position, unsigned int width, unsigned int height, int maxIter)
{
	//code
	unsigned int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	unsigned int j = (blockIdx.y * blockDim.y) + threadIdx.y;

	float u = (float)i / (float)width;
	float v = (float)j / (float)height;

	u = u * 2.0f - 1.0f;
	v = v * 2.0f - 1.0f;

	float x = 0.0f, y = 0.0f;
	double iter = 0.0;
	while(x * x + y * y <= 4.0f && iter < maxIter)
	{
	float xtemp = x * x - y * y + u;
	y = 2.0f * x * y + v;
	x = xtemp;
	iter++;
	}
	iter = iter - log2(log2(x * x + y * y)) + 4.0f;
	float iterf = sin(float(iter) * 0.1f);

	position[(j * width) + i] = make_float4(iterf * 0.3, iterf * 0.5, iterf, 1.0);
}

//Entry Point Function
int WINAPI WinMain(HINSTANCE hInstance, HINSTANCE hPrevInstance, LPSTR lpszCmdLine, int iCmdShow)
{
	//Function Declarations
	int initialize(void);
	void display(void);
	void update(void);
	void uninitialize(void);

	//Variable Declarations
	WNDCLASSEX wndclass;
	HWND hwnd;
	MSG msg;
	TCHAR szAppName[] = TEXT("MyWindow");
	BOOL bDone = FALSE;
	int iRetVal = 0;

	//code
	if (fopen_s(&gpFile, "Log.txt", "w") != 0)
	{
		MessageBox(NULL, TEXT("Creation of Log File Failed. Exiting now!"), TEXT("File I/O Error"), MB_OK | MB_ICONEXCLAMATION);
		exit(0);
	}
	else
		fprintf(gpFile, "Log File Created Successfully!\n");

	//Initialization of WNDCLASSEX structure
	wndclass.cbSize = sizeof(WNDCLASSEX);    //not included in WNDCLASS (only WNDCLASSEX)
	wndclass.style = CS_HREDRAW | CS_VREDRAW | CS_OWNDC;
	wndclass.cbClsExtra = 0;
	wndclass.cbWndExtra = 0;
	wndclass.lpfnWndProc = WndProc;
	wndclass.hInstance = hInstance;
	wndclass.hbrBackground = (HBRUSH)GetStockObject(BLACK_BRUSH);	//Client area color (Canvas)
	wndclass.hIcon = LoadIcon(hInstance, MAKEINTRESOURCE(MYICON));
	wndclass.hCursor = LoadCursor(NULL, IDC_ARROW);
	wndclass.lpszClassName = szAppName;
	wndclass.lpszMenuName = NULL;
	wndclass.hIconSm = LoadIcon(hInstance, MAKEINTRESOURCE(MYICON));    //not included in WNDCLASS

	//Registering Above class
	RegisterClassEx(&wndclass);

	//Creating the window
	hwnd = CreateWindowEx(
		WS_EX_APPWINDOW,																		//Extended style for fullscreen (window that remains above taskbar always)
		szAppName,																				//Name of Wnd class
		TEXT("Shaunak Jotawar OpenGL"),															//Title bar text
		WS_OVERLAPPEDWINDOW | WS_CLIPCHILDREN | WS_CLIPSIBLINGS | WS_VISIBLE,					//style of window 
																								//(3 additional styles ensure fullscreen window stays in focus above all children and sibling windows)
		X / 2 - (WIN_WIDTH / 2),																//x coordinate of top left corner
		Y / 2 - (WIN_HEIGHT / 2),																//y coordinate of top left corner
		WIN_WIDTH,																				//width
		WIN_HEIGHT,																				//height
		NULL,																					//parent window (NULL = Desktop)
		NULL,																					//Handle to Menu (NULL = no menu)
		hInstance,																				//handle to instance
		NULL);																					//Extra info for window

	ghwnd = hwnd;

	//Initialize
	iRetVal = initialize();
	if (iRetVal == -1)
	{
		fprintf(gpFile, "Choose Pixel Format failed\n");
		uninitialize();
	}
	else if (iRetVal == -2)
	{
		fprintf(gpFile, "Set Pixel Format failed\n");
		uninitialize();
	}
	else if (iRetVal == -3)
	{
		fprintf(gpFile, "Create OpenGL context failed\n");
		uninitialize();
	}
	else if (iRetVal == -4)
	{
		fprintf(gpFile, "Making OpenGL as current context failed\n");
		uninitialize();
	}
	else if (iRetVal == -5)
	{
		fprintf(gpFile, "Initialization of GLEW failed\n");
	}
	else
		fprintf(gpFile, "Initialization successful!\n");

	//Show Window
	ShowWindow(hwnd, iCmdShow);

	//Foregrounding and focusing the window
	SetForegroundWindow(hwnd);
	SetFocus(hwnd);

	//Game loop
	while (bDone == FALSE)
	{
		if (PeekMessage(&msg, NULL, 0, 0, PM_REMOVE))
		{
			if (msg.message == WM_QUIT)
				bDone = TRUE;
			else
			{
				TranslateMessage(&msg);
				DispatchMessage(&msg);
			}
		}
		else
		{
			if (gbActiveWindow == TRUE)
			{
				//Rendering the scene
				display();

				//updating the scene
				update();
			}
		}
	}

	uninitialize();
	return ((int)msg.wParam);
}

//Callback Function
LRESULT CALLBACK WndProc(HWND hwnd, UINT iMsg, WPARAM wParam, LPARAM lParam)
{
	//Function Prototypes
	void ToggleFullscreen(void);
	void resize(int, int);

	//code
	switch (iMsg)
	{
	case WM_SETFOCUS:
		gbActiveWindow = TRUE;
		break;

	case WM_KILLFOCUS:
		gbActiveWindow = FALSE;
		break;

	case WM_ERASEBKGND:
		return 0;

	case WM_SIZE:
		resize(LOWORD(lParam), HIWORD(lParam));
		break;

	case WM_CHAR:
		switch (wParam)
		{
		case 'F':
		case 'f':
			ToggleFullscreen();
			break;
		default:
			break;
		}
		break;

	case WM_KEYDOWN:
		switch (wParam)
		{
		case VK_UP:
			iterations += 10;
			iterChanged = TRUE;
			break;
		case VK_DOWN:
			iterations -= 10;
			iterChanged = TRUE;
			break;
		case 27:
			DestroyWindow(hwnd);
			break;
		default:
			break;
		}
		break;

	case WM_CLOSE:
		DestroyWindow(hwnd);
		break;

	case WM_DESTROY:
		PostQuitMessage(0);
		break;

	default:
		break;
	}

	return DefWindowProc(hwnd, iMsg, wParam, lParam);
}

void ToggleFullscreen(void)
{
	//variable declarations
	static DWORD dwStyle;
	static WINDOWPLACEMENT wp;
	MONITORINFO mi;
	LONG left, top, width, height;

	//code
	wp.length = sizeof(WINDOWPLACEMENT);

	if (gbFullscreen == FALSE)
	{
		dwStyle = GetWindowLong(ghwnd, GWL_STYLE);
		if (dwStyle & WS_OVERLAPPEDWINDOW)
		{
			mi.cbSize = sizeof(MONITORINFO);

			if (GetWindowPlacement(ghwnd, &wp) && GetMonitorInfo(MonitorFromWindow(ghwnd, MONITORINFOF_PRIMARY), &mi))
			{
				SetWindowLong(ghwnd, GWL_STYLE, dwStyle & ~WS_OVERLAPPEDWINDOW);

				left = mi.rcMonitor.left;
				top = mi.rcMonitor.top;
				width = mi.rcMonitor.right - mi.rcMonitor.left;
				height = mi.rcMonitor.bottom - mi.rcMonitor.top;

				SetWindowPos(ghwnd, HWND_TOP, left, top, width, height, SWP_NOZORDER | SWP_FRAMECHANGED);
			}

			ShowCursor(FALSE);
			gbFullscreen = TRUE;
		}
	}
	else
	{
		SetWindowLong(ghwnd, GWL_STYLE, dwStyle | WS_OVERLAPPEDWINDOW);
		SetWindowPlacement(ghwnd, &wp);
		SetWindowPos(ghwnd, HWND_TOP, 0, 0, 0, 0, SWP_NOMOVE | SWP_NOOWNERZORDER | SWP_NOZORDER | SWP_NOSIZE | SWP_FRAMECHANGED);

		ShowCursor(TRUE);
		gbFullscreen = FALSE;
	}
}

int initialize(void)
{
	//Function Declarations
	void uninitialize(void);
	void resize(int, int);

	//variable declarations
	PIXELFORMATDESCRIPTOR pfd;
	int iPixelFormatIndex = 0;

	//code
	ZeroMemory(&pfd, sizeof(PIXELFORMATDESCRIPTOR));

	//initializing PIXELFORMATDESCRIPTOR structure
	pfd.nSize = sizeof(PIXELFORMATDESCRIPTOR);
	pfd.nVersion = 1;
	pfd.dwFlags = PFD_DRAW_TO_WINDOW | PFD_SUPPORT_OPENGL | PFD_DOUBLEBUFFER;
	pfd.iPixelType = PFD_TYPE_RGBA;
	pfd.cColorBits = 32;
	pfd.cRedBits = 8;
	pfd.cGreenBits = 8;
	pfd.cBlueBits = 8;
	pfd.cAlphaBits = 8;
	pfd.cDepthBits = 32;

	//get dc
	ghdc = GetDC(ghwnd);

	//Choose Pixel Format
	iPixelFormatIndex = ChoosePixelFormat(ghdc, &pfd);
	if (iPixelFormatIndex == 0)
		return -1;

	//Set the chosen pixel format
	if (SetPixelFormat(ghdc, iPixelFormatIndex, &pfd) == FALSE)
		return -2;

	//Create OpenGL rendering context
	ghrc = wglCreateContext(ghdc);
	if (ghrc == NULL)
		return -3;

	//Make the rendering context as current context
	if (wglMakeCurrent(ghdc, ghrc) == FALSE)
		return -4;

	//GLEW initialization
	if (glewInit() != GLEW_OK)
		return -5;

	//CUDA Initialization
	int dev_count = 0;
	cudaResult = hipGetDeviceCount(&dev_count);
	if (cudaResult != hipSuccess)
	{
		fprintf(gpFile, "Cuda device count failed!!\n");
		uninitialize();
		exit(EXIT_FAILURE);
	}
	else if (dev_count == 0)
	{
		fprintf(gpFile, "No CUDA Supported devices\n");
		uninitialize();
		exit(EXIT_FAILURE);
	}
	fprintf(gpFile, "Number of CUDA Supported devices: %d\n", dev_count);
	
	//select cuda supported device
	hipSetDevice(0);	//selecting the default 0th cuda supported device

	//OGL Initialization
	//Vertex Shader
	const GLchar* vertexShaderSourceCode =
		"#version 460 core" \
		"\n" \
		"in vec4 a_position;" \
		"in vec2 a_texcoord;" \
		"uniform mat4 u_mvpMatrix;" \
		"out vec2 a_texcoord_out;" \
		"void main(void)" \
		"{" \
		"gl_Position = u_mvpMatrix * a_position;" \
		"a_texcoord_out = a_texcoord;" \
		"}";

	GLuint vertexShaderObject = glCreateShader(GL_VERTEX_SHADER);
	glShaderSource(vertexShaderObject, 1, (const GLchar**)&vertexShaderSourceCode, NULL);
	glCompileShader(vertexShaderObject);

	GLint status;
	GLint infoLogLength;
	char *log = NULL;

	glGetShaderiv(vertexShaderObject, GL_COMPILE_STATUS, &status);
	if (status == GL_FALSE)
	{
		glGetShaderiv(vertexShaderObject, GL_INFO_LOG_LENGTH, &infoLogLength);
		if (infoLogLength > 0)
		{
			log = (char *)malloc(infoLogLength);
			if (log != NULL)
			{
				GLsizei written;
				glGetShaderInfoLog(vertexShaderObject, infoLogLength, &written, log);
				fprintf(gpFile, "\n\nVertex Shader Compilation Log : %s\n\n", log);
				free(log);
				log = NULL;
				uninitialize();
			}
		}
	}

	//Fragment Shader
	const GLchar* fragmentShaderSourceCode = 
		"#version 460 core" \
		"\n" \
		"in vec2 a_texcoord_out;" \
		"uniform sampler2D u_textureSampler;" \
		"out vec4 FragColor;" \
		"void main(void)" \
		"{" \
		"FragColor = texture(u_textureSampler, a_texcoord_out);" \
		"}";

	GLuint fragmentShaderObject = glCreateShader(GL_FRAGMENT_SHADER);
	glShaderSource(fragmentShaderObject, 1, (const GLchar**)&fragmentShaderSourceCode, NULL);
	glCompileShader(fragmentShaderObject);

	status = 0;
	infoLogLength = 0;
	log = NULL;

	glGetShaderiv(fragmentShaderObject, GL_COMPILE_STATUS, &status);
	if (status == GL_FALSE)
	{
		glGetShaderiv(fragmentShaderObject, GL_INFO_LOG_LENGTH, &infoLogLength);
		if (infoLogLength > 0)
		{
			log = (char*)malloc(infoLogLength);
			if (log != NULL)
			{
				GLsizei written;
				glGetShaderInfoLog(fragmentShaderObject, infoLogLength, &written, log);
				fprintf(gpFile, "\n\nFragment Shader Compilation Log : %s", log);
				free(log);
				log = NULL;
				uninitialize();
			}
		}
	}

	//Shader Program object
	shaderProgramObject = glCreateProgram();
	glAttachShader(shaderProgramObject, vertexShaderObject);
	glAttachShader(shaderProgramObject, fragmentShaderObject);
	
	glBindAttribLocation(shaderProgramObject, SDJ_ATTRIBUTE_POSITION, "a_position");
	glBindAttribLocation(shaderProgramObject, SDJ_ATTRIBUTE_TEXTURE0, "a_texcoord");

	glLinkProgram(shaderProgramObject);


	//ERROR Checking
	status = 0;
	infoLogLength = 0;
	log = NULL;

	glGetProgramiv(shaderProgramObject, GL_LINK_STATUS, &status);
	if (status == GL_FALSE)
	{
		glGetProgramiv(shaderProgramObject, GL_INFO_LOG_LENGTH, &infoLogLength);
		if (infoLogLength > 0)
		{
			log = (char*)malloc(infoLogLength);
			if (log != NULL)
			{
				GLsizei written;
				glGetProgramInfoLog(shaderProgramObject, infoLogLength, &written, log);
				fprintf(gpFile, "\n\nShader Program Link log : %s", log);
				free(log);
				log = NULL;
				uninitialize();
			}
		}
	}
	
	//post link steps
	mvpMatrixUniform = glGetUniformLocation(shaderProgramObject, "u_mvpMatrix");
	textureSamplerUniform = glGetUniformLocation(shaderProgramObject, "u_textureSampler");

	//declarations of vertex data arrays
	const GLfloat position[]
	{
		1.0f, 1.0f, 0.0f,
		-1.0f, 1.0f, 0.0f,
		-1.0f, -1.0f, 0.0f,
		1.0f, -1.0f, 0.0f,
	};

	const GLfloat texCoords[] =
	{
		1.0f, 1.0f,
		0.0f, 1.0f,
		0.0f, 0.0f,
		1.0f, 0.0f,
	};

	//vao and vbo related code
	glGenVertexArrays(1, &vao);
	glBindVertexArray(vao);

	glGenBuffers(1, &vbo);
	glBindBuffer(GL_ARRAY_BUFFER, vbo);
		glBufferData(GL_ARRAY_BUFFER, sizeof(position), position, GL_STATIC_DRAW);
		glVertexAttribPointer(SDJ_ATTRIBUTE_POSITION, 3, GL_FLOAT, GL_FALSE, 0, NULL);
		glEnableVertexAttribArray(SDJ_ATTRIBUTE_POSITION);
	glBindBuffer(GL_ARRAY_BUFFER, 0);

	//vertex buffer object for ___texcoords___
	glGenBuffers(1, &vbo_texcoord);
	glBindBuffer(GL_ARRAY_BUFFER, vbo_texcoord);
		glBufferData(GL_ARRAY_BUFFER, sizeof(texCoords), texCoords, GL_STATIC_DRAW);
		glVertexAttribPointer(SDJ_ATTRIBUTE_TEXTURE0, 2, GL_FLOAT, GL_FALSE, 0, NULL);
		glEnableVertexAttribArray(SDJ_ATTRIBUTE_TEXTURE0);
	glBindBuffer(GL_ARRAY_BUFFER, 0);

	glBindVertexArray(0);

	//allocate memory on GPU
	cudaResult = hipMalloc((void **)&pPos, MYARRAYSIZE * sizeof(float4));
	if (cudaResult != hipSuccess)
	{
		fprintf(gpFile, "hipMalloc() failed for pPos!!\n");
		uninitialize();
		exit(EXIT_FAILURE);
	}

	//allocate memory
	pColor = (vec4 *)malloc(MYARRAYSIZE * sizeof(vec4));
	if (pColor == NULL)
	{
		fprintf(gpFile, "malloc() failed for pColor!!\n");
		uninitialize();
		exit(EXIT_FAILURE);
	}

	//Depth enabling and clear color functions
	glClearDepth(1.0f);
	glEnable(GL_DEPTH_TEST);
	glEnable(GL_TEXTURE_2D);
	glDepthFunc(GL_LEQUAL);

	glClearColor(0.0f, 0.0f, 0.0f, 1.0f);

	//initializing orthographic projection matrix
	perspectiveProjectionMatrix = mat4::identity();
	resize(WIN_WIDTH, WIN_HEIGHT);

	return 0;
}

void resize(int width, int height)
{
	//code
	if (height == 0)
		height = 1;

	glViewport(0, 0, (GLsizei)width, (GLsizei)height);

	perspectiveProjectionMatrix = vmath::perspective(
		45.0, (GLfloat)width / (GLfloat)height, 0.1f, 100.0f
	);
}

void display(void)
{
	//function prototypes
	void sineWave(unsigned int, unsigned int, float);
	void uninitialize(void);

	//variable declarations
	size_t numBytes;

	//code
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	//use shader program object
	glUseProgram(shaderProgramObject);

	//transformations
	mat4 translationMatrix = mat4::identity();
	mat4 modelViewMatrix = mat4::identity();
	mat4 modelViewProjectionMatrix = mat4::identity();
	
	translationMatrix = translate(0.0f, 0.0f, -1.0f);
	modelViewMatrix = translationMatrix;
	modelViewProjectionMatrix = perspectiveProjectionMatrix * modelViewMatrix;
	glUniformMatrix4fv(mvpMatrixUniform, 1, GL_FALSE, modelViewProjectionMatrix);

	if (iterChanged)
	{
		//cuda code
		//map cuda graphics resource
		cudaResult = hipGraphicsMapResources(1, &graphicsResource, 0);
		if (cudaResult != hipSuccess)
		{
			fprintf(gpFile, "hipGraphicsMapResources() failed!!\n");
			uninitialize();
			exit(EXIT_FAILURE);
		}

		//ask cuda to give mapped pointer of mapped resource
		cudaResult = hipGraphicsResourceGetMappedPointer((void **)&pPos, &numBytes, graphicsResource);
		if (cudaResult != hipSuccess)
		{
			fprintf(gpFile, "hipGraphicsResourceGetMappedPointer() failed!!\n");
			uninitialize();
			exit(EXIT_FAILURE);
		}

		//call cuda sineWave Kernel
		dim3 block = dim3(8, 8, 1);
		dim3 grid = dim3(meshWidth / block.x, meshHeight / block.y, 1);

		mandelbrotKernel <<<grid, block>>> (pPos, meshWidth, meshHeight, iterations);

		//unmap given mapped pointer to use on cpu
		cudaResult = hipGraphicsUnmapResources(1, &graphicsResource, 0);
		if (cudaResult != hipSuccess)
		{
			fprintf(gpFile, "hipGraphicsUnmapResources() failed!!\n");
			uninitialize();
			exit(EXIT_FAILURE);
		}

		glPixelStorei(GL_UNPACK_ALIGNMENT, 1);

		glGenTextures(1, &texture_Mandelbrot);
		glBindTexture(GL_TEXTURE_2D, texture_Mandelbrot);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);

		glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, meshWidth, meshHeight, 0, GL_RGBA, GL_UNSIGNED_BYTE, pPos);
		glTexEnvf(GL_TEXTURE_ENV, GL_TEXTURE_ENV_MODE, GL_REPLACE);

		glBindTexture(GL_TEXTURE_2D, 0);

		iterChanged = FALSE;
	}

	//mandelbrot texture binding
	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_2D, texture_Mandelbrot);
	glUniform1i(textureSamplerUniform, 0);
		
	//draw arrays
	glBindVertexArray(vao);
	glDrawArrays(GL_TRIANGLE_FAN, 0, 4);
	glBindVertexArray(0);

	//unuse the shader program object
	glUseProgram(0);

	SwapBuffers(ghdc);
}

void update(void)
{
	//code
}

void uninitialize(void)
{
	//Function declarations
	void ToggleFullscreen(void);

	//code
	if (gbFullscreen)
	{
		ToggleFullscreen();
	}

	//deletion and uninitialization of vbo
	if (vbo_gpu)
	{
		if (graphicsResource)
		{
			hipGraphicsUnregisterResource(graphicsResource);
			graphicsResource = NULL;
		}
		glDeleteBuffers(1, &vbo_gpu);
		vbo_gpu = 0;
	}

	if (vbo)
	{
		glDeleteBuffers(1, &vbo);
		vbo = 0;
	}

	//deletion and uninitialization of vao
	if (vao)
	{
		glDeleteVertexArrays(1, &vao);
		vao = 0;
	}

	//shader uninitialization
	if (shaderProgramObject)
	{
		glUseProgram(shaderProgramObject);
		GLsizei numAttachedShaders;
		glGetProgramiv(shaderProgramObject, GL_ATTACHED_SHADERS, &numAttachedShaders);
		GLuint *shaderObjects = NULL;
		shaderObjects = (GLuint *)malloc(numAttachedShaders * sizeof(GLuint));

		//filling empty buffer with shader objects
		glGetAttachedShaders(shaderProgramObject, numAttachedShaders, &numAttachedShaders, shaderObjects);

		for (GLsizei i = 0; i < numAttachedShaders; i++)
		{
			glDetachShader(shaderProgramObject, shaderObjects[i]);
			glDeleteShader(shaderObjects[i]);
			shaderObjects[i] = 0;
		}

		free(shaderObjects);
		shaderObjects = NULL;
		glUseProgram(0);
		glDeleteProgram(shaderProgramObject);
		shaderProgramObject = 0;
	}

	if (wglGetCurrentContext() == ghrc)
	{
		wglMakeCurrent(NULL, NULL);
		fprintf(gpFile, "Current context initialized to NULL\n");
	}

	if (ghrc)
	{
		wglDeleteContext(ghrc);
		ghrc = NULL;
		fprintf(gpFile, "Current context deleted\n");
	}

	if (ghdc)
	{
		ReleaseDC(ghwnd, ghdc);
		ghdc = NULL;
		fprintf(gpFile, "Device context released\n");
	}

	if (ghwnd)
	{
		fprintf(gpFile, "Window destroyed\n");
		DestroyWindow(ghwnd);
		ghwnd = NULL;
	}

	if (gpFile)
	{
		fprintf(gpFile, "Log File closed successfully!\n");
		fclose(gpFile);
		gpFile = NULL;
	}

	exit(0);
}
